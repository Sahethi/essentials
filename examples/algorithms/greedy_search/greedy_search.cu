#include "hip/hip_runtime.h"
#include <gunrock/algorithms/greedy_search.hxx>
#include "greedy_search_cpu.hxx"  // Reference implementation
#include <fstream>

using namespace std;
using namespace gunrock;
using namespace memory;

vector<double> calculate_distances(int x, int y){
  ifstream infile("/content/essentials/examples/algorithms/greedy_search/points.txt");
  string line;
  vector<double> euclidean;
  while (getline(infile, line)) {
      istringstream iss(line);
      int a, b;
      if (!(iss >> a >> b)) { break; } // error
      double dist = sqrt(pow(x-a, 2) + pow(y-b, 2));
      euclidean.push_back(dist);
  }

  return euclidean;
}

void test_greedy_search(int num_arguments, char** argument_array, vector<double> euclidean_distances) {
  if (num_arguments != 4) {
     cerr << "usage: ./bin/<program-name> filename.mtx x y" <<  endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  csr_t csr;
   string filename = argument_array[1];

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
     cerr << "Unknown file format: " << filename <<  endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation
  srand(time(NULL));
  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<vertex_t> nodes(n_vertices);
  vertex_t single_source = 0;
  for(int j = 0; j < n_vertices; j++){
    nodes[j] = j;
  }
  single_source = 0;
  cout << "Single Source = " << single_source <<  endl;
    // --
    // GPU Run

    /// An example of how one can use std::shared_ptr to allocate memory on the
    /// GPU, using a custom deleter that automatically handles deletion of the
    /// memory.
    // std::shared_ptr<weight_t> distances(
    //     allocate<weight_t>(n_vertices * sizeof(weight_t)),
    //     deleter_t<weight_t>());
    // std::shared_ptr<vertex_t> predecessors(
    //     allocate<vertex_t>(n_vertices * sizeof(vertex_t)),
    //     deleter_t<vertex_t>());

    thrust::device_vector<weight_t> distances(n_vertices);
    thrust::device_vector<vertex_t> predecessors(n_vertices);

    float gpu_elapsed = 0.0f;
    int num_runs = 5;

    for (auto i = 0; i < num_runs; i++)
      gpu_elapsed += gunrock::greedy_search::run(G, single_source, euclidean_distances, distances.data().get(),
                                        predecessors.data().get(), nodes.data().get());

    gpu_elapsed /= num_runs;

    // --
    // CPU Run

    thrust::host_vector<weight_t> h_distances(n_vertices);
    thrust::host_vector<vertex_t> h_predecessors(n_vertices);
    thrust::host_vector<vertex_t> h_nodes(n_vertices);

    float cpu_elapsed = greedy_search_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
        csr, single_source, euclidean_distances, h_distances.data(), h_predecessors.data(), h_nodes.data());

    int n_errors =
        util::compare(distances.data().get(), h_distances.data(), n_vertices);

    // --
    // Log + Validate


    cout << "\nGPU Elapsed Time : " << gpu_elapsed << " (ms)" <<  endl;
    cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" <<  endl;
    cout << "Number of errors : " << n_errors <<  endl;
  
}

int main(int argc, char** argv) {
  // int x = 10, y = 4;
  vector<double> euclidean;
  int x = atoi(argv[2]);
  int y = atoi(argv[3]);
  euclidean = calculate_distances(10, 4);
  for (auto i = euclidean.begin(); i != euclidean.end(); ++i)
    cout << *i << " ";
  cout<<"\n";
  test_greedy_search(argc, argv, euclidean);
}
